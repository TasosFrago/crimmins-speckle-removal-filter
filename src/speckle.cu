#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define CHECK_CUDA(call)                                                       \
  do {                                                                         \
    hipError_t err = (call);                                                  \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__,            \
              hipGetErrorString(err));                                        \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

#define TIME_FUNC(func, time)                                                  \
  do {                                                                         \
    time = clock();                                                            \
    (func);                                                                    \
    time = (clock() - time) / CLOCKS_PER_SEC;                                  \
    printf("Time taken: %lfs\n", time);                                        \
  } while (0)

#define TIME_FUNC3(func, time, res)                                            \
  do {                                                                         \
    time = clock();                                                            \
    res = (func);                                                              \
    time = (clock() - time) / CLOCKS_PER_SEC;                                  \
    printf("Time taken: %lfs\n", time);                                        \
  } while (0)

#define SWAP(a, b)                                                             \
  do {                                                                         \
    uint8_t *__tmp = (a);                                                      \
    (a) = (b);                                                                 \
    (b) = __tmp;                                                               \
  } while (0);

#define ITERATIONS 1

uint8_t dark_pass_logic(uint8_t a, uint8_t b, uint8_t c);
uint8_t light_pass_logic(uint8_t a, uint8_t b, uint8_t c);

struct PassDescr {
  int dx, dy;
  uint8_t (*pass_logic_func)(uint8_t, uint8_t, uint8_t);
};

const static struct PassDescr passes[] = {
    {1, 0, dark_pass_logic},  // Horizontal (dx, dy) = (1,0)
    {0, 1, dark_pass_logic},  // Vertical (dx, dy) = (0,1)
    {1, 1, dark_pass_logic},  // Diagonal (dx, dy) = (1,1)
    {1, -1, dark_pass_logic}, // Other Diagonal (dx, dy) = (1,-1)

    {1, 0, light_pass_logic},  // Horizontal (dx, dy) = (1,0)
    {0, 1, light_pass_logic},  // Vertical (dx, dy) = (0,1)
    {1, 1, light_pass_logic},  // Diagonal (dx, dy) = (1,1)
    {1, -1, light_pass_logic}, // Other Diagonal (dx, dy) = (1,-1)
};

int fwriteImage(const char *output_path, uint8_t *image, uint32_t image_size);
void image_validator(uint8_t *image1, uint8_t *image2, uint32_t width,
                     uint32_t height);

/* Device versions of your logic functions */
__device__ uint8_t dark_pass_logic_dev(uint8_t a, uint8_t b, uint8_t c) {
  if (a >= b + 2)
    b++;
  if (a > b && b <= c)
    b++;
  if (c > b && b <= a)
    b++;
  if (c >= b + 2)
    b++;
  return b;
}

__device__ uint8_t light_pass_logic_dev(uint8_t a, uint8_t b, uint8_t c) {
  if (a <= b - 2)
    b--;
  if (a < b && b >= c)
    b--;
  if (c < b && b >= a)
    b--;
  if (c <= b - 2)
    b--;
  return b;
}

/* Single CUDA kernel per pass */
__global__ void crimmins_pass_kernel(uint8_t *out, const uint8_t *in, int width,
                                     int height, int dx, int dy, int is_light) {
  int x = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int y = blockIdx.y * blockDim.y + threadIdx.y + 1;

  if (x >= width - 1 || y >= height - 1)
    return;

  uint8_t a = in[(y - dy) * width + (x - dx)];
  uint8_t b = in[y * width + x];
  uint8_t c = in[(y + dy) * width + (x + dx)];

  b = is_light ? light_pass_logic_dev(a, b, c) : dark_pass_logic_dev(a, b, c);

  out[y * width + x] = b;
}

/* Host API: allocate device buffers, launch passes, swap, copy back */
float crimmings_speckle_removal_filter_cuda(uint8_t *h_image, uint32_t width,
                                            uint32_t height,
                                            uint8_t iterations) {
  size_t numPixels = (size_t)(width * height);
  size_t bufBytes = numPixels * sizeof(uint8_t);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  uint8_t *d_buf1 = NULL;
  uint8_t *d_buf2 = NULL;
  CHECK_CUDA(hipMalloc((void **)&d_buf1, bufBytes));
  CHECK_CUDA(hipMalloc((void **)&d_buf2, bufBytes));

  /* copy host→device into d_buf2 */
  CHECK_CUDA(hipMemcpy(d_buf1, h_image, bufBytes, hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_buf2, h_image, bufBytes, hipMemcpyHostToDevice));

  /* block/grid dims */
  dim3 block = {16, 16, 1};
  dim3 grid = {(uint32_t)((width + block.x - 3) / block.x),
               (uint32_t)((height + block.y - 3) / block.y), 1};
  printf("grid.x = %d, grid.y = %d\n", grid.x, grid.y);

  uint8_t *d_in = d_buf2;
  uint8_t *d_out = d_buf1;

  for (int iter = 0; iter < iterations; iter++) {
    for (int p = 0; p < 8; p++) {
      int is_light = (passes[p].pass_logic_func == light_pass_logic);

      crimmins_pass_kernel<<<grid, block>>>(
          d_out, d_in, width, height, passes[p].dx, passes[p].dy, is_light);
      // CHECK_CUDA(cudaGetLastError());

      SWAP(d_out, d_in);
    }
  }
  // if ((iterations * 8) % 2 == 1) {
  // 	SWAP(d_out, d_in);
  // }
  CHECK_CUDA(hipMemcpy(h_image, d_out, bufBytes, hipMemcpyDeviceToHost));

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float time = 0;
  hipEventElapsedTime(&time, start, stop);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  time = time * 1e-3f;

  hipFree(d_buf1);
  hipFree(d_buf2);
  return time;
}

uint8_t dark_pass_logic(uint8_t a, uint8_t b, uint8_t c) {
  if (a >= b + 2)
    b++;
  if (a > b && b <= c)
    b++;
  if (c > b && b <= a)
    b++;
  if (c >= b + 2)
    b++;
  return b;
}

uint8_t light_pass_logic(uint8_t a, uint8_t b, uint8_t c) {
  if (a <= b - 2)
    b--;
  if (a < b && b >= c)
    b--;
  if (c < b && b >= a)
    b--;
  if (c <= b - 2)
    b--;
  return b;
}

// Serial Implementation

void pass_func(uint8_t *image, uint8_t *tmp_image, uint32_t width,
               uint32_t height, int dx, int dy,
               uint8_t (*pass_logic_func)(uint8_t, uint8_t, uint8_t)) {
  for (int y = 1; y < height - 1; y++) {
    uint8_t *row_a = tmp_image + (y - dy) * width;
    uint8_t *row_b = tmp_image + y * width;
    uint8_t *row_c = tmp_image + (y + dy) * width;
    uint8_t *row_out = image + y * width;

    for (int x = 1; x < width - 1; x++) {
      uint8_t a = row_a[x - dx];
      uint8_t b = row_b[x];
      uint8_t c = row_c[x + dx];

      b = pass_logic_func(a, b, c);

      row_out[x] = b;
    }
  }
}

void crimmings_speckle_removal_filter(uint8_t *image, uint32_t width,
                                      uint32_t height, uint8_t iterations) {
  uint8_t *tmp_image = (uint8_t *)malloc(width * height);
  assert(tmp_image != NULL && "Failed allocating temp image buffer");

  memcpy(tmp_image, image, width * height);
  for (int i = 0; i < iterations; i++) {
    for (int p = 0; p < 8; p++) {
      pass_func(image, tmp_image, width, height, passes[p].dx, passes[p].dy,
                passes[p].pass_logic_func);
      SWAP(image, tmp_image);
    }
  }
  // if ( (iterations * 8) % 2 == 1 ) {
  // 	memcpy(image, tmp_image, width*height);
  // }
  free(tmp_image);
}

int main(int argc, char *argv[]) {
  if (argc < 4) {
    fprintf(stderr, "Not enough input arguments given.\n");
    fprintf(stderr,
            "USAGE: %s path/to/input/image [width] [height] [optional num of "
            "passes]\n",
            argv[0]);
    return -1;
  }

  const char *input_path = argv[1];
  const uint32_t width = atoi(argv[2]);
  const uint32_t height = atoi(argv[3]);
  const uint8_t iterations = (argc >= 4) ? atoi(argv[4]) : ITERATIONS;

  size_t image_s = width * height;

  FILE *infd = fopen(input_path, "rb");
  if (!infd) {
    fprintf(stderr, "Error opening the input image\n");
    return -2;
  }

  uint8_t *image_ser = (uint8_t *)malloc(image_s);
  uint8_t *image_par = (uint8_t *)malloc(image_s);
  if (!image_ser || !image_par) {
    fprintf(stderr, "Error allocating image buffer");
    fclose(infd);
    return -3;
  }
  if (fread(image_ser, 1, image_s, infd) != image_s) {
    fprintf(stderr, "Error reading input image\n");
    fclose(infd);
    free(image_ser);
    free(image_par);
    return -4;
  }
  fclose(infd);
  memcpy(image_par, image_ser, image_s);

  printf("Serial algorithm\n");
  double time_ser;
  TIME_FUNC(
      crimmings_speckle_removal_filter(image_ser, width, height, iterations),
      time_ser);

  //==========================================
  int res = fwriteImage("out_ser.raw", image_ser, image_s);
  if (res) {
    free(image_ser);
    free(image_par);
    return -1;
  }
  //==========================================

  printf("Parallel algorithm\n");
  double time_par, time_par_cuda;
  // double time_par_cuda = crimmings_speckle_removal_filter_cuda(image_par,
  // width, 						 height, iterations);
  TIME_FUNC3(crimmings_speckle_removal_filter_cuda(image_par, width, height,
                                                   iterations),
             time_par, time_par_cuda);
  printf("Time cuda taken: %lfs\n", time_par_cuda);

  //==========================================
  res = fwriteImage("out_par.raw", image_par, image_s);
  if (res) {
    free(image_ser);
    free(image_par);
    return -5;
  }
  //==========================================
  printf("Speedup: %.4f\n", (time_ser / time_par));

  printf("\nValidating ser with par\n");
  image_validator(image_ser, image_par, width, height);

  free(image_ser);
  free(image_par);
  printf("Filtering complete!!!\n");

  return 0;
}

void image_validator(uint8_t *image1, uint8_t *image2, uint32_t width,
                     uint32_t height) {
  int error_cnt = 0;
  int idx = 0;
  FILE *resf = fopen("out.log", "w");
  assert(resf != NULL);
  int sum = 0;

  for (int y = 0; y < height; y++) {
    for (int x = 0; x < width; x++) {
      idx = y * width + x;
      if (image1[idx] != image2[idx]) {
        error_cnt++;
        sum += abs(image1[idx] - image2[idx]);
        fprintf(resf, "Error on pixel (%d,\t%d),\t with %d\n", x, y,
                (image1[idx] - image2[idx]));
      }
    }
  }
  fclose(resf);

  double avg_err = (sum != 0) ? (float)sum / error_cnt : 0;
  printf("Error cnt = %d / %d,\t avg_err  = %.4f\n", error_cnt, width * height,
         avg_err);
}

int fwriteImage(const char *output_path, uint8_t *image, uint32_t image_size) {
  assert(output_path != NULL && "Incorect output file");
  FILE *outfd = fopen(output_path, "wb");
  if (!outfd) {
    fprintf(stderr, "Error writing output file %s.\n", output_path);
    return -1;
  }
  fwrite(image, 1, image_size, outfd);
  fclose(outfd);
  return 0;
}
